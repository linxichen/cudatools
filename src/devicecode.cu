#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <iomanip>
#include <string>
#include "../include/common.hpp"
#include "../include/devicecode.cuh"

////////////////////////////////////////
//
/// Interpolation Stuff
//
////////////////////////////////////////
// Linear interpolation
__host__ __device__
double linear_interp(double x, double x_left, double x_right, double f_left, double f_right) {
	if (abs(x_left-x_right)<1e-10) {
		return f_left;
	} else if (x_left > x_right) {
		return -1200981025976;
	} else {
		return f_left + (f_right-f_left)/(x_right-x_left)*(x-x_left);
	};
};

// Bilinear interpolation
__host__ __device__
double bilinear_interp(double x, double y, double x_left, double x_right, double y_low, double y_high, double f_leftlow, double f_lefthigh, double f_rightlow, double f_righthigh) {
	double f_low = linear_interp(x,x_left,x_right,f_leftlow,f_rightlow);
	double f_high = linear_interp(x,x_left,x_right,f_lefthigh,f_righthigh);
	return linear_interp(y,y_low,y_high,f_low,f_high);
};

// This function converts index to subscripts like ind2sub in MATLAB
__host__ __device__
void ind2sub(int length_size, int* siz_vec, int index, int* subs) {
// Purpose:		Converts index to subscripts. i -> [i_1, i_2, ..., i_n]
//
// Input:		length_size = # of coordinates, i.e. how many subscripts you are getting
// 				siz_vec = vector that stores the largest coordinate value for each subscripts. Or the dimensions of matrices
// 				index = the scalar index
//
// Ouput:		subs = the vector stores subscripts
	int done = 0;
	for (int i=length_size-1; i>=0; i--) {
		// Computer the cumulative dimension
		int cumdim = 1;
		for (int j=0; j<=i-1; j++) {
			cumdim *= siz_vec[j];
		};
		int temp_sub = (index - done)/cumdim;
		subs[i] = temp_sub;
		done += temp_sub*cumdim;
	};
};

// This function fit a valuex x to a increasing grid X of size n.
// The largest value on grid X that is smaller than x is returned ("left grid point" is returned).
__host__ __device__
int fit2grid(const double x, const int n, const double* X) {
	if (x < X[0]) {
		return 0;
	} else if (x >= X[n-1]) {
		return n-1;
	} else {
		int left=0; int right=n-1; int mid=(n-1)/2;
		while(right-left>1) {
			mid = (left + right)/2;
			if (X[mid]==x) {
				return mid;
			} else if (X[mid]<x) {
				left = mid;
			} else {
				right = mid;
			};
		};
		return left;
	}
};

// This function fit a valuex x to a increasing grid X of size n.
// The largest value on grid X that is smaller than x is returned ("left grid point" is returned).
// grid is accessed with stride s. we are looking at j = 1:n X[stride+j*n]
__host__ __device__
int fit2grid(const double x, const int n, const double* X, const int stride) {
	if (x < X[stride+0*n]) {
		return 0;
	} else if (x >= X[stride+(n-1)*n]) {
		return n-1;
	} else {
		int left=0; int right=n-1; int mid=(n-1)/2;
		while(right-left>1) {
			mid = (left + right)/2;
			if (X[stride+mid*n]==x) {
				return mid;
			} else if (X[stride+mid*n]<x) {
				left = mid;
			} else {
				right = mid;
			};
		};
		return left;
	}
}

// This function fit a valuex x to a "even" grid X of size n. Even means equi-distance among grid points.
// The largest value on grid X that is smaller than x is returned ("left grid point" is returned).
__host__ __device__
int fit2evengrid(const double x, const int n, const double min, const double max) {
	if (x <= min) return 0;
	if (x >= max) return n-1;
	double step = (max-min)/(n-1);
	return floor((x-min)/step);
};

// This function fit a valuex x to a grid X of size n. For std::vector like stuff
// The largest value on grid X that is smaller than x is returned ("left grid point" is returned).


////////////////////////////////////////
//
// Chebyshev Toolset
//
////////////////////////////////////////
// Evaluate Chebychev polynomial of any degree
__host__ __device__
double chebypoly(const int p, const double x) {
	switch (p) {
		case 0: // 0-th order Chebyshev Polynomial
			return 1;
		case 1:
			return x;
		case 2:
			return 2*x*x - 1;
		case 3:
			return 4*x*x*x - 3*x;
	}

	// When p>=4, apply the recurrence relation
	double lag1 = 4*x*x*x -3*x;
	double lag2 = 2*x*x - 1;
	double lag0;
	int distance = p - 3;
	while (distance >= 1) {
		lag0 = 2*x*lag1 - lag2;
		lag2 = lag1;
		lag1 = lag0;
		distance--;
	};
	return lag0;
};

// Evaluate Chebychev polynomial of any degree
__host__ __device__
int chebyroots(const int p, double* roots) {
	for (int i=0; i<p; i++) {
		double stuff = p - 0.5 - 1*i;
		roots[i] = cos(M_PI*(stuff)/(p));
	};

	// Account for the fact that cos(pi/2) is not exactly zeros
	if (p%2) {
		roots[(p-1)/2] = 0;
	};
	return 0;
};

// Evaluate Chebychev approximation of any degree
__host__ __device__
double chebyeval(int p, double x, double* coeff) {
	// Note that coefficient vector has p+1 values
	double sum = 0;
	for (int i=0; i<=p; i++) {
		sum += coeff[i]*chebypoly(i,x);
	};
	return sum;
};

// Eval multi-dimensional Chebyshev tensor basis
// y = sum T_pi(x_i), i = 1,2,...p
__host__ __device__
double chebyeval_multi (const int n_var, double* x, int* size_vec,int* temp_subs, double* coeff) {
	// Note size_vec's elements are p+1 for each var
	int tot_deg = 1;
	for (int i = 0; i < n_var; i++) {
		tot_deg *= (size_vec[i]); // Note there's p+1 coeffs
	};

	double eval = 0;
	for (int index = 0; index < tot_deg; index++) {
		// Perform ind2sub to get current degrees for each var
		ind2sub(n_var, size_vec, index, temp_subs);

		// Find the values at current degrees
		double temp = 1;
		for (int i = 0; i < n_var; i++) {
			// printf("%i th subscript is %i\n",i,temp_subs[i]);
			temp *= chebypoly(temp_subs[i],x[i]);
		};

		// Add to the eval
		eval += (coeff[index]*temp);
	};
	return eval;
};



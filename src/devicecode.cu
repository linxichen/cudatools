#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <iomanip>
#include <string>
#include "../include/common.hpp"
#include "../include/devicecode.cuh"

////////////////////////////////////////
//
/// Interpolation Stuff
//
////////////////////////////////////////
// Linear interpolation
__host__ __device__
double linear_interp(double x, double x_left, double x_right, double f_left, double f_right) {
	if (abs(x_left-x_right)<1e-10) {
		return f_left;
	} else if (x_left > x_right) {
		return -1200981025976;
	} else {
		return f_left + (f_right-f_left)/(x_right-x_left)*(x-x_left);
	};
};

// Bilinear interpolation
__host__ __device__
double bilinear_interp(double x, double y, double x_left, double x_right, double y_low, double y_high, double f_leftlow, double f_lefthigh, double f_rightlow, double f_righthigh) {
	double f_low = linear_interp(x,x_left,x_right,f_leftlow,f_rightlow);
	double f_high = linear_interp(x,x_left,x_right,f_lefthigh,f_righthigh);
	return linear_interp(y,y_low,y_high,f_low,f_high);
};

// This function converts index to subscripts like ind2sub in MATLAB
__host__ __device__
void ind2sub(int length_size, int* siz_vec, int index, int* subs) {
// Purpose:		Converts index to subscripts. i -> [i_1, i_2, ..., i_n]
//
// Input:		length_size = # of coordinates, i.e. how many subscripts you are getting
// 				siz_vec = vector that stores the largest coordinate value for each subscripts. Or the dimensions of matrices
// 				index = the scalar index
//
// Ouput:		subs = the vector stores subscripts
	int done = 0;
	for (int i=length_size-1; i>=0; i--) {
		// Computer the cumulative dimension
		int cumdim = 1;
		for (int j=0; j<=i-1; j++) {
			cumdim *= siz_vec[j];
		};
		int temp_sub = (index - done)/cumdim;
		subs[i] = temp_sub;
		done += temp_sub*cumdim;
	};
};

// This function fit a valuex x to a increasing grid X of size n.
// The largest value on grid X that is smaller than x is returned ("left grid point" is returned).
__host__ __device__
int fit2grid(const double x, const int n, const double* X) {
	if (x < X[0]) {
		return 0;
	} else if (x >= X[n-1]) {
		return n-1;
	} else {
		int left=0; int right=n-1; int mid=(n-1)/2;
		while(right-left>1) {
			mid = (left + right)/2;
			if (X[mid]==x) {
				return mid;
			} else if (X[mid]<x) {
				left = mid;
			} else {
				right = mid;
			};
		};
		return left;
	}
};

// This function fit a valuex x to a increasing grid X of size n.
// The largest value on grid X that is smaller than x is returned ("left grid point" is returned).
// grid is accessed with stride s. we are looking at j = 1:n X[stride+j*n]
__host__ __device__
int fit2grid(const double x, const int n, const double* X, const int stride) {
	if (x < X[stride+0*n]) {
		return 0;
	} else if (x >= X[stride+(n-1)*n]) {
		return n-1;
	} else {
		int left=0; int right=n-1; int mid=(n-1)/2;
		while(right-left>1) {
			mid = (left + right)/2;
			if (X[stride+mid*n]==x) {
				return mid;
			} else if (X[stride+mid*n]<x) {
				left = mid;
			} else {
				right = mid;
			};
		};
		return left;
	}
}

// This function fit a valuex x to a "even" grid X of size n. Even means equi-distance among grid points.
// The largest value on grid X that is smaller than x is returned ("left grid point" is returned).
__host__ __device__
int fit2evengrid(const double x, const int n, const double min, const double max) {
	if (x <= min) return 0;
	if (x >= max) return n-1;
	double step = (max-min)/(n-1);
	return floor((x-min)/step);
};


/// This function fit a valuex x to a grid X of size n.
/// For std::vector like stuff
/// The largest value on grid X that is smaller than x is
/// returned ("left grid point" is returned).
template <class T>
int fit2grid(const double x, const T X) {
	int n = X.size();
	return fit2grid( x, n, X);
};

////////////////////////////////////////
//
// Chebyshev Toolset
//
////////////////////////////////////////
// Evaluate Chebychev polynomial of any degree
__host__ __device__
double chebypoly(const int p, const double x) {
	switch (p) {
		case 0: // 0-th order Chebyshev Polynomial
			return 1;
		case 1:
			return x;
		case 2:
			return 2*x*x - 1;
		case 3:
			return 4*x*x*x - 3*x;
	}

	// When p>=4, apply the recurrence relation
	double lag1 = 4*x*x*x -3*x;
	double lag2 = 2*x*x - 1;
	double lag0;
	int distance = p - 3;
	while (distance >= 1) {
		lag0 = 2*x*lag1 - lag2;
		lag2 = lag1;
		lag1 = lag0;
		distance--;
	};
	return lag0;
};

// Evaluate Chebychev polynomial of any degree
__host__ __device__
int chebyroots(const int p, double* roots) {
	for (int i=0; i<p; i++) {
		double stuff = p - 0.5 - 1*i;
		roots[i] = cos(M_PI*(stuff)/(p));
	};

	// Account for the fact that cos(pi/2) is not exactly zeros
	if (p%2) {
		roots[(p-1)/2] = 0;
	};
	return 0;
};

// Evaluate Chebychev approximation of any degree
__host__ __device__
double chebyeval(int p, double x, double* coeff) {
	// Note that coefficient vector has p+1 values
	double sum = 0;
	for (int i=0; i<=p; i++) {
		sum += coeff[i]*chebypoly(i,x);
	};
	return sum;
};

// Eval multi-dimensional Chebyshev tensor basis
// y = sum T_pi(x_i), i = 1,2,...p
__host__ __device__
double chebyeval_multi (const int n_var, double* x, int* size_vec,int* temp_subs, double* coeff) {
	// Note size_vec's elements are p+1 for each var
	int tot_deg = 1;
	for (int i = 0; i < n_var; i++) {
		tot_deg *= (size_vec[i]); // Note there's p+1 coeffs
	};

	double eval = 0;
	for (int index = 0; index < tot_deg; index++) {
		// Perform ind2sub to get current degrees for each var
		ind2sub(n_var, size_vec, index, temp_subs);

		// Find the values at current degrees
		double temp = 1;
		for (int i = 0; i < n_var; i++) {
			// printf("%i th subscript is %i\n",i,temp_subs[i]);
			temp *= chebypoly(temp_subs[i],x[i]);
		};

		// Add to the eval
		eval += (coeff[index]*temp);
	};
	return eval;
};

////////////////////////////////////////////////////////////////////////
///
/// Nonlinear Solver stuff
///
////////////////////////////////////////////////////////////////////////
template <class T>
__host__ __device__
double newton_bracket(T func, double x1, double x2, double x0) {
// Purpose: Tries to find a root for function named func.
// Its first derivative is given by func.prime().
// It is assumed that func(x1) and func(x2) are different in sign so
// a root exists within. x0 is the guess.

	const int newton_maxiter = 100;
	const double newton_tol = 1e-3;
	// Checking the bounds: they need to make sense.
	// Or sometimes the bounds are solutions.
	double f1 = func(x1);
	double f2 = func(x2);
	if (f1*f2>0) return -5179394.1; // The different sign assumption violated!
	if (f1 == 0) return x1;
	if (f2 == 0) return x2;

	// Orient the search so that f(xl) < 0
	double xl, xh;
	if (f1 < 0.0) {
		xl = x1;
		xh = x2;
	} else {
		xh = x1;
		xl = x2;
	};

	// Initialize guess and other things
	double rts = x0;
	double dxold = abs(x2-x1);
	double dx = dxold;
	double f = func(rts);
	double df = func.prime(rts);

	for (int iter = 0; iter < newton_maxiter; iter++) {
		if (
			( ((rts-xh)*df-f)*((rts-xl)*df-f) > 0.0 )   ||	// Bisect if Newton step out of range
			( abs(2.0*f) > abs(dxold*df)  ) // ... or step not decreasing fast enough
		)
		{
			dxold = dx;
			dx = 0.5*(xh-xl);
			rts += dxold; // undo the newton step
			rts = xl + dx;
			if (xl == rts) return rts;
		} else {
			// If newton step is okay
			dxold = dx;
			dx = f/df;
			double temp = rts;
			rts -= dx;
			if (temp==rts) return rts;
		};

		// Check for convergence
		if ( abs(dx)/(1+abs(rts+dx)) < newton_tol ) return rts;

		// Compute new f and df for next iteration
		f = func(rts);
		df = func.prime(rts);

		// Maintain the bracket
		if (f < 0.0) {
			xl = rts;
		} else {
			xh = rts;
		};
	};

	return -51709394.2;
};

template <class T>
__host__ __device__
double newton(T func, const double x1, const double x2, double x0) {
	const int newton_maxiter = 20;
	const double newton_tol = 1e-4;
	// Initialize guess and other things
	double x_old = x0;
	double x = x0;
	double f1 = func(x1);
	double f2 = func(x2);
	if (f1==0) return x1;
	if (f2==0) return x2;
	for (int iter = 0; iter < newton_maxiter; iter++) {
		x = x_old - func(x)/func.prime(x);

		// Pull back if outside of support
		if (x<=x1) {
			return -51709394.2;
		};
		if (x>=x2) {
			return -51709394.2;
		};

		// Check for convergence
		if ( (abs(x-x_old)/(1+abs(x_old))<newton_tol) && (abs(func(x)) < newton_tol) ) {
			return x;
		} else {
			x_old = x;
		};
	};
	return -51709394.2;
};

////////////////////////////////////////
//
// Some tools for simulation
//
////////////////////////////////////////

// a quick and dirty exclusive scan to turn
// markov transition matrix into CDF matrix
__host__ __device__
void pdf2cdf(double* P, size_t n, double* CDF) {
	for (int i_now = 0; i_now < n; i_now++) {
		CDF[i_now+0*n] = 0;
		for (unsigned int i_tmr = 1; i_tmr < n; i_tmr++) {
			CDF[i_now+i_tmr*n] = P[i_now+(i_tmr-1)*n] + CDF[i_now+(i_tmr-1)*n];
		};
	};
};

// draw from a n-state discrete distribution, wit aug-CDF
// given as a n-by-n array begins with 0 and then cumsum(PDF(1:end)).
// e.g at i_now = 0, PDF(i_now,:) = [0.2 0.3 0.5], then CDF[i_now,:] = [0 0.2 0.5]
// also given a random number in [0,1] for inverse CDF.
/// highly recomment to use with pdf2cdf function.
__host__ __device__
int markovdiscrete(int i_now, double* CDF, size_t n, double u) {
	for (unsigned int i_tmr = 1; i_tmr < n; i_tmr++) {
		if ( CDF[i_now+i_tmr*n] > u ) {
			return i_tmr-1;
		}
	};
	return n-1;
};

__host__ __device__
void markovsimul(int T, double* CDF, int n, double* u, int init, int* sim) {
	sim[0] = init;
	for (int t = 1; t < T; t++) {
		sim[t] = markovdiscrete(sim[t-1],CDF,n,u[t]);
	};
};
